/*
Matrix Multiplication, with help from cuBLASLt
*/
#include <assert.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <type_traits> // std::bool_constant
// llmc internal imports
#include "cublas_common.h"
#include "cuda_common.h"

#define PY_SSIZE_T_CLEAN
#include <Python.h>
#include <hipblaslt.h>
#include <string>

// Global variables

// Wrapper around hipblasLtMatmul that is meant to support everything we need in
// llm.c https://docs.nvidia.com/cuda/cublas/#cublasltmatmul
#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t error = call;                                                  \
    if (error != hipSuccess) {                                                \
      std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << ": "     \
                << hipGetErrorString(error) << std::endl;                     \
      throw std::runtime_error("CUDA error");                                  \
    }                                                                          \
  } while (0)

#define CUBLAS_CHECK(call)                                                     \
  do {                                                                         \
    hipblasStatus_t status = call;                                              \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
      std::cerr << "cuBLAS error in " << __FILE__ << ":" << __LINE__ << ": "   \
                << cublasGetStatusString(status) << std::endl;                 \
      throw std::runtime_error("cuBLAS error");                                \
    }                                                                          \
  } while (0)

#define SAFE_GET_ARRAY_POINTER(arr, ptr_var)                                   \
  data_attr = PyObject_GetAttrString(arr, "data");                             \
  if (!data_attr) {                                                            \
    PyErr_SetString(PyExc_AttributeError,                                      \
                    "Array object has no attribute 'data'");                   \
    return NULL;                                                               \
  }                                                                            \
  ptr = PyLong_AsVoidPtr(data_attr);                                           \
  Py_DECREF(data_attr);                                                        \
  if (PyErr_Occurred()) {                                                      \
    return NULL;                                                               \
  }                                                                            \
  ptr_var = static_cast<float *>(ptr);

static int init_cublas() {
  if (hipblasLtCreate(&cublaslt_handle) != HIPBLAS_STATUS_SUCCESS) {
    PyErr_SetString(PyExc_RuntimeError, "Failed to create cuBLASLt handle");
    return -1;
  }
  if (hipMalloc(&cublaslt_workspace, cublaslt_workspace_size) != hipSuccess) {
    hipblasLtDestroy(cublaslt_handle);
    cublaslt_handle = nullptr;
    PyErr_SetString(PyExc_RuntimeError,
                    "Failed to allocate cuBLASLt workspace");
    return -1;
  }
  return 0;
}

static void cleanup_cublas() {
  if (cublaslt_workspace) {
    hipFree(cublaslt_workspace);
    cublaslt_workspace = nullptr;
  }
  if (cublaslt_handle) {
    hipblasLtDestroy(cublaslt_handle);
    cublaslt_handle = nullptr;
  }
}
void *get_array_pointer(PyObject *arr, const char *name) {
  std::cout << "Entering get_array_pointer for " << name << std::endl;

  if (!arr) {
    std::cerr << name << " is NULL" << std::endl;
    PyErr_SetString(PyExc_TypeError, "Array object is NULL");
    return nullptr;
  }

  std::cout << "Array object type: " << Py_TYPE(arr)->tp_name << std::endl;

  if (!PyObject_HasAttrString(arr, "data")) {
    std::cerr << name << " has no 'data' attribute" << std::endl;
    PyErr_SetString(PyExc_AttributeError,
                    "Array object has no attribute 'data'");
    return nullptr;
  }

  PyObject *data_attr = PyObject_GetAttrString(arr, "data");
  if (!data_attr) {
    std::cerr << "Failed to get 'data' attribute for " << name << std::endl;
    PyErr_Print();
    return nullptr;
  }

  std::string data_type_name = Py_TYPE(data_attr)->tp_name;
  std::cout << "Type of data attribute for " << name << ": " << data_type_name
            << std::endl;

  // Handle MemoryPointer object
  if (data_type_name.find("MemoryPointer") != std::string::npos) {
    PyObject *ptr_attr = PyObject_GetAttrString(data_attr, "ptr");
    if (!ptr_attr) {
      std::cerr << "Failed to get 'ptr' attribute from MemoryPointer for "
                << name << std::endl;
      Py_DECREF(data_attr);
      PyErr_Print();
      return nullptr;
    }

    void *ptr = PyLong_AsVoidPtr(ptr_attr);
    Py_DECREF(ptr_attr);
    Py_DECREF(data_attr);

    if (PyErr_Occurred()) {
      std::cerr << "Error occurred while getting pointer for " << name
                << std::endl;
      PyErr_Print();
      return nullptr;
    }

    if (!ptr) {
      std::cerr << "Got NULL pointer for " << name << std::endl;
      PyErr_SetString(PyExc_ValueError, "Got NULL pointer from array data");
      return nullptr;
    }

    std::cout << "Successfully got pointer for " << name << ": " << ptr
              << std::endl;
    return ptr;
  } else {
    std::cerr << "Unexpected data attribute type for " << name << ": "
              << data_type_name << std::endl;
    Py_DECREF(data_attr);
    PyErr_SetString(PyExc_TypeError, "Unexpected data attribute type");
    return nullptr;
  }
}

void matmul_cublaslt(floatX *d, const floatX *a, const floatX *b,
                     const floatX *bias, int m, int n, int k,
                     hipStream_t stream = 0, bool transA = true,
                     bool transB = false, int batch_count = 0,
                     size_t strideA = 0, size_t strideB = 0,
                     size_t strideOut = 0, bool accumulate = false,
                     bool backward = false) {
  std::cout << "Entering matmul_cublaslt" << std::endl;
  std::cout << "Dimensions: m=" << m << ", n=" << n << ", k=" << k << std::endl;
  std::cout << "Pointers: d=" << d << ", a=" << a << ", b=" << b
            << ", bias=" << bias << std::endl;
  std::cout << "Other params: transA=" << transA << ", transB=" << transB
            << ", batch_count=" << batch_count << ", accumulate=" << accumulate
            << ", backward=" << backward << std::endl;

  NVTX_RANGE_FN();
  bool has_bias = (bias != NULL);
  std::cout << "has_bias: " << has_bias << std::endl;

  // check alignment
  if (((uintptr_t)a % 16) != 0 || ((uintptr_t)b % 16) != 0 ||
      ((uintptr_t)d % 16) != 0 || (has_bias && ((uintptr_t)bias % 16) != 0)) {
    throw std::runtime_error("All cuBLASLt pointers must be aligned!");
  }
  std::cout << "Alignment check passed" << std::endl;

  // create the operation descriptor
  hipblasLtMatmulDesc_t operationDesc = nullptr;
  hipblasLtMatrixLayout_t ALayout = nullptr, BLayout = nullptr,
                         CLayout = nullptr, DLayout = nullptr;
  hipblasLtMatmulPreference_t preference = nullptr;

  try {
    std::cout << "Creating matmul descriptor" << std::endl;
    CUBLAS_CHECK(
        hipblasLtMatmulDescCreate(&operationDesc, cublas_compute, HIP_R_32F));
    std::cout << "Matmul descriptor created" << std::endl;

    // Set matrix operation attributes
    hipblasOperation_t opNoTranspose = HIPBLAS_OP_N;
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;
    std::cout << "Setting matrix operation attributes" << std::endl;
    CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA,
        (transA) ? &opTranspose : &opNoTranspose, sizeof(opTranspose)));
    CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB,
        (transB) ? &opTranspose : &opNoTranspose, sizeof(opNoTranspose)));
    std::cout << "Matrix operation attributes set" << std::endl;

    // define matrix layouts
    std::cout << "Creating matrix layouts" << std::endl;
    if (transA) {
      CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&ALayout, CUBLAS_LOWP, k, m, k));
    } else {
      CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&ALayout, CUBLAS_LOWP, m, k, m));
    }
    if (transB) {
      CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&BLayout, CUBLAS_LOWP, n, k, n));
    } else {
      CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&BLayout, CUBLAS_LOWP, k, n, k));
    }
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(
        &CLayout, (sizeof(floatX) == 1) ? HIP_R_16BF : CUBLAS_LOWP, m, n, m));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&DLayout, CUBLAS_LOWP, m, n, m));
    std::cout << "Matrix layouts created" << std::endl;

    // create a preference handle
    std::cout << "Creating preference handle" << std::endl;
    CUBLAS_CHECK(hipblasLtMatmulPreferenceCreate(&preference));
    CUBLAS_CHECK(hipblasLtMatmulPreferenceSetAttribute(
        preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
        &cublaslt_workspace_size, sizeof(cublaslt_workspace_size)));
    std::cout << "Preference handle created" << std::endl;

    // find a suitable algorithm
    std::cout << "Finding suitable algorithm" << std::endl;
    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristic;
    CUBLAS_CHECK(hipblasLtMatmulAlgoGetHeuristic(
        cublaslt_handle, operationDesc, ALayout, BLayout, CLayout, DLayout,
        preference, 1, &heuristic, &returnedResults));

    if (returnedResults == 0) {
      throw std::runtime_error("No suitable cuBLASLt algorithm found");
    }
    std::cout << "Suitable algorithm found" << std::endl;

    // set whether to accumulate or not
    std::cout << "Setting alpha and beta" << std::endl;
    const float alpha = 1.0f, beta = accumulate ? 1.0f : 0.0f;
    std::cout << "Alpha and beta set: alpha=" << alpha << ", beta=" << beta
              << std::endl;

    // call the matmul
    std::cout << "Calling hipblasLtMatmul" << std::endl;
    CUBLAS_CHECK(hipblasLtMatmul(cublaslt_handle, operationDesc, &alpha, a,
                                ALayout, b, BLayout, &beta, d, CLayout, d,
                                DLayout, &heuristic.algo, cublaslt_workspace,
                                cublaslt_workspace_size, stream));

    std::cout << "hipblasLtMatmul completed successfully" << std::endl;
  } catch (const std::exception &e) {
    std::cerr << "Error in matmul_cublaslt: " << e.what() << std::endl;
    // Clean up resources
    if (preference)
      hipblasLtMatmulPreferenceDestroy(preference);
    if (operationDesc)
      hipblasLtMatmulDescDestroy(operationDesc);
    if (ALayout)
      hipblasLtMatrixLayoutDestroy(ALayout);
    if (BLayout)
      hipblasLtMatrixLayoutDestroy(BLayout);
    if (CLayout)
      hipblasLtMatrixLayoutDestroy(CLayout);
    if (DLayout)
      hipblasLtMatrixLayoutDestroy(DLayout);
    throw; // re-throw the exception
  }

  // Clean up resources
  hipblasLtMatmulPreferenceDestroy(preference);
  hipblasLtMatmulDescDestroy(operationDesc);
  hipblasLtMatrixLayoutDestroy(ALayout);
  hipblasLtMatrixLayoutDestroy(BLayout);
  hipblasLtMatrixLayoutDestroy(CLayout);
  hipblasLtMatrixLayoutDestroy(DLayout);

  CUDA_CHECK(hipGetLastError());
  std::cout << "Exiting matmul_cublaslt" << std::endl;
}
extern "C" void matmul_forward_cublaslt(floatX *out, floatX *inp,
                                        floatX *weight, floatX *bias, int B,
                                        int T, int C, int OC,
                                        hipStream_t stream = 0) {
  std::cout << "Entering matmul_forward_cublaslt" << std::endl;
  std::cout << "Dimensions: B=" << B << ", T=" << T << ", C=" << C
            << ", OC=" << OC << std::endl;
  std::cout << "Pointers: out=" << out << ", inp=" << inp
            << ", weight=" << weight << ", bias=" << bias << std::endl;

  try {
    if (cublaslt_handle == nullptr) {
      throw std::runtime_error("cublaslt_handle is not initialized");
    }
    if (cublaslt_workspace == nullptr) {
      throw std::runtime_error("cublaslt_workspace is not initialized");
    }
    matmul_cublaslt(out, weight, inp, bias, OC, B * T, C, stream, true, false,
                    0, 0, 0, 0, false);
  } catch (const std::exception &e) {
    std::cerr << "Exception in matmul_forward_cublaslt: " << e.what()
              << std::endl;
    throw;
  }

  std::cout << "Exiting matmul_forward_cublaslt" << std::endl;
}

static PyObject *py_matmul_forward_cublaslt(PyObject *self, PyObject *args) {
  PyObject *output, *input, *weights, *bias;
  long B, T, C, out_shape;
  PyObject *stream_obj;

  if (!PyArg_ParseTuple(args, "OOOOllllO", &output, &input, &weights, &bias, &B,
                        &T, &C, &out_shape, &stream_obj)) {
    PyErr_Print();
    return NULL;
  }

  std::cout << "Parsed arguments in py_matmul_forward_cublaslt" << std::endl;
  std::cout << "B: " << B << ", T: " << T << ", C: " << C
            << ", out_shape: " << out_shape << std::endl;

  float *output_ptr, *input_ptr, *weights_ptr, *bias_ptr;

  output_ptr = static_cast<float *>(get_array_pointer(output, "output"));
  if (!output_ptr)
    return NULL;

  input_ptr = static_cast<float *>(get_array_pointer(input, "input"));
  if (!input_ptr)
    return NULL;

  weights_ptr = static_cast<float *>(get_array_pointer(weights, "weights"));
  if (!weights_ptr)
    return NULL;

  bias_ptr = static_cast<float *>(get_array_pointer(bias, "bias"));
  if (!bias_ptr)
    return NULL; // Handle CUDA stream

  hipStream_t hip_stream = nullptr;
  if (PyObject_HasAttrString(stream_obj, "ptr")) {
    PyObject *ptr_attr = PyObject_GetAttrString(stream_obj, "ptr");
    if (ptr_attr) {
      hip_stream = reinterpret_cast<hipStream_t>(PyLong_AsVoidPtr(ptr_attr));
      Py_DECREF(ptr_attr);
    } else {
      PyErr_Clear(); // Clear the attribute error
    }
  }

  if (hip_stream == nullptr) {
    std::cout << "Using default stream (nullptr)" << std::endl;
  } else {
    std::cout << "Using provided stream: " << hip_stream << std::endl;
  }
  std::cout << "Extracted CUDA stream" << std::endl;

  try {
    matmul_forward_cublaslt(output_ptr, input_ptr, weights_ptr, bias_ptr, B, T,
                            C, out_shape, hip_stream);
  } catch (const std::exception &e) {
    PyErr_SetString(PyExc_RuntimeError, e.what());
    return NULL;
  }

  std::cout << "Completed py_matmul_forward_cublaslt" << std::endl;

  Py_RETURN_NONE;
}

// Module method definitions
static PyMethodDef LlmcMethods[] = {
    {"matmul_forward_cublaslt", py_matmul_forward_cublaslt, METH_VARARGS,
     "Perform matrix multiplication using cuBLAS LT"},
    {NULL, NULL, 0, NULL} // Sentinel
};

// Module definition
static struct PyModuleDef llmcmodule = {PyModuleDef_HEAD_INIT, "llmc", NULL, -1,
                                        LlmcMethods};

static void llmc_free(void *unused) { cleanup_cublas(); }

PyMODINIT_FUNC PyInit_llmc(void) {
  PyObject *m;

  m = PyModule_Create(&llmcmodule);
  if (m == NULL)
    return NULL;

  if (init_cublas() < 0) {
    Py_DECREF(m);
    return NULL;
  }

  if (PyModule_AddFunctions(m, LlmcMethods) < 0) {
    Py_DECREF(m);
    return NULL;
  }

  if (PyModule_AddObject(
          m, "__cleanup__",
          PyCapsule_New((void *)llmc_free, "__cleanup__", NULL)) < 0) {
    Py_DECREF(m);
    return NULL;
  }

  return m;
}
