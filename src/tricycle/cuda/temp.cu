
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <hipblaslt.h>

int main() {
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS initialization failed\n");
        return 1;
    }
    printf("cuBLAS initialized successfully\n");
    hipblasDestroy(handle);
    return 0;
}
